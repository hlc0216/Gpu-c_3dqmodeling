#include "hip/hip_runtime.h"
/************ The program is writed by Lun Ruan, 2018.10***********************/
/*******3D Modeling for pure qP wave equation from Xu,2015************/

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>
#include <malloc.h>
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>
//#include <hip/device_functions.h>

#include "array_new.h"
#include "read_write.h"
#include "GPU_kernel.h"
#include "CPU_function.h"

#define M 10
#define eps  2.22e-17
#define Block_Sizex 8
#define Block_Sizey 8
#define Block_Sizez 8


void modeling3d(int nx, int ny, int nz, int nt, int ntsnap, float dx, float dy, float dz, float dt, int pml, int snapflag, int sx, int sy, int sz, 
                float *vp, float *epsilon, float *delta, float *source, float *wavelet, float *record, float *dlr,float *ddlr, float *dtb, float *ddtb, 
				float *dfb, float *ddfb, float *c, float *c2, const char *snap_file)							
{
		//time-assuming
		clock_t starttime, endtime;
		float timespent;
		starttime = clock();

		int device_num;
		hipGetDeviceCount(&device_num);
		if(device_num > 0)
			hipSetDevice(0);
		else
			hipSetDevice(0);

		float uxMax, uyMax, uzMax, uxyzMax;
		int i,j,l,k;
		char snapname[100], snapname_S[100], snapxzname[100], snapyzname[100],snapxyname[100],
			snapSxzname[100], snapSyzname[100],snapSxyname[100];
		
		dim3 grid((ny+Block_Sizey-1)/Block_Sizey, (nx+Block_Sizex-1)/Block_Sizex, (nz+Block_Sizez-1)/Block_Sizez);
		dim3 block(Block_Sizey, Block_Sizex, Block_Sizez);
		
		//allocate host memory
		float	*snap = array1d((nx-2*pml)*(ny-2*pml)*(nz-2*pml)), *snapxz = array1d((nx-2*pml)*(nz-2*pml)), 
				*snapyz = array1d((ny-2*pml)*(nz-2*pml)), *snapxy = array1d((ny-2*pml)*(nx-2*pml)),
				*snapS = array1d(nx*ny*nz), *snapSxz = array1d(nx*nz), *snapSyz = array1d(ny*nz), *snapSxy = array1d(ny*nx),
				*h_ux = array1d(nx*ny*nz), *h_uy = array1d(nx*ny*nz), *h_uz = array1d(nx*ny*nz),*h_u2 = array1d(nx*ny*nz);
		
		/******* allocate device memory *****/
		float	*d_vp, *d_epsilon,*d_delta,*d_c,*d_c2,*d_dlr,*d_ddlr,*d_dtb,*d_ddtb,*d_dfb,
				*d_ddfb, *d_source, *S, *u1, *u2, *u3, *ux, *uy, *uz,*d_record, *d_u,				
				*wl11, *wl12, *wl13, *wl21, *wl31, *wl32, *wl33, *pl1,*pl2,*pl3,
				*wr11, *wr12, *wr13, *wr21, *wr31, *wr32, *wr33, *pr1,*pr2,*pr3,
				*wt11, *wt12, *wt13, *wt21, *wt31, *wt32, *wt33, *pt1,*pt2,*pt3,
				*wb11, *wb12, *wb13, *wb21, *wb31, *wb32, *wb33, *pb1,*pb2,*pb3,
				*wf11, *wf12, *wf13, *wf21, *wf31, *wf32, *wf33, *pf1,*pf2,*pf3,
				*wba11, *wba12, *wba13, *wba21, *wba31, *wba32, *wba33, *pba1,*pba2,*pba3;

		/*打印cpu参数*/
		for(i=0;i<nt;i++){
			printf("wavelet=%4.3f ",wavelet[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("dlr=%4.3f ",dlr[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("ddlr=%4.3f ",ddlr[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("dtb=%4.3f ",dtb[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("ddtb=%4.3f ",ddtb[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("dfb=%4.3f ",dfb[i]);
		}
		printf("\n");
		for(i=0;i<pml;i++){
			printf("ddfb=%4.3f ",ddfb[i]);
		}
		printf("\n");
		for(i=(M/2)*(M/2+1)+1;i<(M/2)*(M/2+1)+6;i++){
			printf("c2=%4.3f ",c2[i]);
		}
		printf("\n");
		
		hipMalloc(&d_vp, nx*ny*nz*sizeof(float));
		hipMalloc(&d_epsilon, nx*ny*nz*sizeof(float));
		hipMalloc(&d_delta, nx*ny*nz*sizeof(float));
		hipMalloc(&d_c, (M/2+1)*(M/2+1)*sizeof(float));
		hipMalloc(&d_c2, (M/2+1)*(M/2+1)*sizeof(float));
		hipMalloc(&d_dlr,pml*sizeof(float));
		hipMalloc(&d_ddlr, pml*sizeof(float));
		hipMalloc(&d_dtb, pml*sizeof(float));
		hipMalloc(&d_ddtb, pml*sizeof(float));
		hipMalloc(&d_dfb, pml*sizeof(float));
		hipMalloc(&d_ddfb, pml*sizeof(float));
		hipMalloc(&d_source, nx*ny*nz*sizeof(float));
		hipMalloc(&S, nx*ny*nz*sizeof(float));
		hipMalloc(&u1, nx*ny*nz*sizeof(float));
		hipMalloc(&u2, nx*ny*nz*sizeof(float));
		hipMalloc(&u3, nx*ny*nz*sizeof(float));
		hipMalloc(&ux, nx*ny*nz*sizeof(float));
		hipMalloc(&uy, nx*ny*nz*sizeof(float));
		hipMalloc(&uz, nx*ny*nz*sizeof(float));
	//	hipMalloc(&d_record, (nx-2*pml)*(ny-2*pml)*nt*sizeof(float));
	//	hipMalloc(&d_u, (nx-2*pml)*(ny-2*pml)*(nz-2*pml)*sizeof(float));
	
		hipMalloc(&wr11, pml*ny*nz*sizeof(float));
		hipMalloc(&wr12, pml*ny*nz*sizeof(float));
		hipMalloc(&wr13, pml*ny*nz*sizeof(float));
		hipMalloc(&wr21, pml*ny*nz*sizeof(float));
		hipMalloc(&wr31, pml*ny*nz*sizeof(float));
		hipMalloc(&wr32, pml*ny*nz*sizeof(float));
		hipMalloc(&wr33, pml*ny*nz*sizeof(float));
		hipMalloc(&pr1, pml*ny*nz*sizeof(float));
		hipMalloc(&pr2, pml*ny*nz*sizeof(float));
		hipMalloc(&pr3, pml*ny*nz*sizeof(float));

		hipMalloc(&wl11, pml*ny*nz*sizeof(float));
		hipMalloc(&wl12, pml*ny*nz*sizeof(float));
		hipMalloc(&wl13, pml*ny*nz*sizeof(float));
		hipMalloc(&wl21, pml*ny*nz*sizeof(float));
		hipMalloc(&wl31, pml*ny*nz*sizeof(float));
		hipMalloc(&wl32, pml*ny*nz*sizeof(float));
		hipMalloc(&wl33, pml*ny*nz*sizeof(float));
		hipMalloc(&pl1, pml*ny*nz*sizeof(float));
		hipMalloc(&pl2, pml*ny*nz*sizeof(float));
		hipMalloc(&pl3, pml*ny*nz*sizeof(float));

		hipMalloc(&wt11, pml*nx*ny*sizeof(float));
		hipMalloc(&wt12, pml*nx*ny*sizeof(float));
		hipMalloc(&wt13, pml*nx*ny*sizeof(float));
		hipMalloc(&wt21, pml*nx*ny*sizeof(float));
		hipMalloc(&wt31, pml*nx*ny*sizeof(float));
		hipMalloc(&wt32, pml*nx*ny*sizeof(float));
		hipMalloc(&wt33, pml*nx*ny*sizeof(float));
		hipMalloc(&pt1, pml*nx*ny*sizeof(float));
		hipMalloc(&pt2, pml*nx*ny*sizeof(float));
		hipMalloc(&pt3, pml*nx*ny*sizeof(float));

		hipMalloc(&wb11, pml*nx*ny*sizeof(float));
		hipMalloc(&wb12, pml*nx*ny*sizeof(float));
		hipMalloc(&wb13, pml*nx*ny*sizeof(float));
		hipMalloc(&wb21, pml*nx*ny*sizeof(float));
		hipMalloc(&wb31, pml*nx*ny*sizeof(float));
		hipMalloc(&wb32, pml*nx*ny*sizeof(float));
		hipMalloc(&wb33, pml*nx*ny*sizeof(float));
		hipMalloc(&pb1, pml*nx*ny*sizeof(float));
		hipMalloc(&pb2, pml*nx*ny*sizeof(float));
		hipMalloc(&pb3, pml*nx*ny*sizeof(float));
		
		hipMalloc(&wf11, pml*nx*nz*sizeof(float));
		hipMalloc(&wf12, pml*nx*nz*sizeof(float));
		hipMalloc(&wf13, pml*nx*nz*sizeof(float));
		hipMalloc(&wf21, pml*nx*nz*sizeof(float));
		hipMalloc(&wf31, pml*nx*nz*sizeof(float));
		hipMalloc(&wf32, pml*nx*nz*sizeof(float));
		hipMalloc(&wf33, pml*nx*nz*sizeof(float));
		hipMalloc(&pf1, pml*nx*nz*sizeof(float));
		hipMalloc(&pf2, pml*nx*nz*sizeof(float));
		hipMalloc(&pf3, pml*nx*nz*sizeof(float));

		hipMalloc(&wba11, pml*nx*nz*sizeof(float));
		hipMalloc(&wba12, pml*nx*nz*sizeof(float));
		hipMalloc(&wba13, pml*nx*nz*sizeof(float));
		hipMalloc(&wba21, pml*nx*nz*sizeof(float));
		hipMalloc(&wba31, pml*nx*nz*sizeof(float));
		hipMalloc(&wba32, pml*nx*nz*sizeof(float));
		hipMalloc(&wba33, pml*nx*nz*sizeof(float));
		hipMalloc(&pba1, pml*nx*nz*sizeof(float));
		hipMalloc(&pba2, pml*nx*nz*sizeof(float));
		hipMalloc(&pba3, pml*nx*nz*sizeof(float));
		
		
	//intialized memory
	/*	hipMemset(S, 0, nx*ny*nz*sizeof(float));
		hipMemset(u1, 0, nx*ny*nz*sizeof(float));
		hipMemset(u2, 0, nx*ny*nz*sizeof(float));
		hipMemset(u3, 0, nx*ny*nz*sizeof(float));
		hipMemset(ux, 0, nx*ny*nz*sizeof(float));
		hipMemset(uy, 0, nx*ny*nz*sizeof(float));
		hipMemset(uz, 0, nx*ny*nz*sizeof(float));

		hipMemset(wr11, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr12, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr13, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr21, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr31, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr32, 0, pml*ny*nz*sizeof(float));
		hipMemset(wr33, 0, pml*ny*nz*sizeof(float));
		hipMemset(pr1, 0, pml*ny*nz*sizeof(float));
		hipMemset(pr2, 0, pml*ny*nz*sizeof(float));
		hipMemset(pr3, 0, pml*ny*nz*sizeof(float));

		hipMemset(wl11, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl12, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl13, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl21, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl31, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl32, 0, pml*ny*nz*sizeof(float));
		hipMemset(wl33, 0, pml*ny*nz*sizeof(float));
		hipMemset(pl1, 0, pml*ny*nz*sizeof(float));
		hipMemset(pl2, 0, pml*ny*nz*sizeof(float));
		hipMemset(pl3, 0, pml*ny*nz*sizeof(float));

		hipMemset(wt11, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt12, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt13, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt21, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt31, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt32, 0, pml*nx*ny*sizeof(float));
		hipMemset(wt33, 0, pml*nx*ny*sizeof(float));
		hipMemset(pt1, 0, pml*nx*ny*sizeof(float));
		hipMemset(pt2, 0, pml*nx*ny*sizeof(float));
		hipMemset(pt3, 0, pml*nx*ny*sizeof(float));

		hipMemset(wb11, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb12, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb13, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb21, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb31, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb32, 0, pml*nx*ny*sizeof(float));
		hipMemset(wb33, 0, pml*nx*ny*sizeof(float));
		hipMemset(pb1, 0, pml*nx*ny*sizeof(float));
		hipMemset(pb2, 0, pml*nx*ny*sizeof(float));
		hipMemset(pb3, 0, pml*nx*ny*sizeof(float));
		
		hipMemset(wf11, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf12, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf13, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf21, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf31, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf32, 0, pml*nx*nz*sizeof(float));
		hipMemset(wf33, 0, pml*nx*nz*sizeof(float));
		hipMemset(pf1, 0, pml*nx*nz*sizeof(float));
		hipMemset(pf2, 0, pml*nx*nz*sizeof(float));
		hipMemset(pf3, 0, pml*nx*nz*sizeof(float));

		hipMemset(wba11, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba12, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba13, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba21, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba31, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba32, 0, pml*nx*nz*sizeof(float));
		hipMemset(wba33, 0, pml*nx*nz*sizeof(float));
		hipMemset(pba1, 0, pml*nx*nz*sizeof(float));
		hipMemset(pba2, 0, pml*nx*nz*sizeof(float));
		hipMemset(pba3, 0, pml*nx*nz*sizeof(float));
	*/	
		hipMemcpy(d_vp, vp, nx*ny*nz*sizeof(float), hipMemcpyHostToDevice);	
		hipMemcpy(d_epsilon, epsilon,  nx*ny*nz*sizeof(float), hipMemcpyHostToDevice);		
		hipMemcpy(d_delta, delta, nx*ny*nz*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_c, c, (M/2+1)*(M/2+1)*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_c2, c2, (M/2+1)*(M/2+1)*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_source, source, nx*ny*nz*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dlr, dlr,  pml*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_ddlr, ddlr,  pml*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dtb, dtb,   pml*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_ddtb, ddtb,  pml*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_dfb, dfb,   pml*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(d_ddfb, ddfb,  pml*sizeof(float), hipMemcpyHostToDevice);
		
		for(k=0;k<nt;k++)
   		{
  			if(k%100==0)
				printf("nt = %d\n",k);
				
			grad<<<grid,block>>>(u2, ux, uy, uz, d_c, nx, ny, nz, dx, dy, dz);
			
			hipMemcpy(h_ux, ux, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_uy, uy, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);
			hipMemcpy(h_uz, uz, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);

			uxMax = absMaxval(h_ux, nx, ny, nz);
			uyMax = absMaxval(h_uy, nx, ny, nz);
			uzMax = absMaxval(h_uz, nx, ny, nz);
			uxyzMax = max(uxMax, uyMax);
			uxyzMax = max(uxyzMax, uzMax);
			//打印uxyzMax
			printf("uxyzMax=%4.3f\n",uxyzMax);
			//calculating S operators
  	  		scalar_operator<<<grid,block>>>(uxyzMax, ux, uy, uz, d_epsilon, d_delta, S, nx, ny, nz);
  	 	    //打印S GPU端(见GPU_kernel.cu)
  	  		//calculating wavefield using FD method
  	  		wavefield_update<<<grid,block>>>(d_c, d_c2, d_dlr, d_ddlr, d_dtb, d_ddtb, d_dfb, d_ddfb, d_epsilon,d_delta,
  	                                    	d_vp, dx, dy, dz, dt, nx, ny, nz, pml, sz, ux, uy, uz, u1, u3, u2, S,
											wl11, wl12, wl13, wl21, wl31, wl32, wl33, pl1, pl2, pl3,
											wr11, wr12, wr13, wr21, wr31, wr32, wr33, pr1, pr2, pr3,
											wt11, wt12, wt13, wt21, wt31, wt32, wt33, pt1, pt2, pt3,
											wb11, wb12, wb13, wb21, wb31, wb32, wb33, pb1, pb2, pb3,
											wf11, wf12, wf13, wf21, wf31, wf32, wf33, pf1, pf2, pf3,
											wba11, wba12, wba13, wba21, wba31, wba32, wba33, pba1, pba2, pba3);
  	  		addsource<<<grid,block>>>(d_source, wavelet[k], u3, nx, ny, nz);
			
  	 		exchange<<<grid,block>>>(nx, ny, nz, pml, u1, u2, u3, 
									wl11, wl12, wl13, wl31, wl32, wl33, pl1, pl2, pl3,
									wr11, wr12, wr13, wr31, wr32, wr33, pr1, pr2, pr3,
									wt11, wt12, wt13, wt31, wt32, wt33, pt1, pt2, pt3,
									wb11, wb12, wb13, wb31, wb32, wb33, pb1, pb2, pb3,
									wf11, wf12, wf13, wf31, wf32, wf33, pf1, pf2, pf3,
									wba11, wba12, wba13, wba31, wba32, wba33, pba1, pba2, pba3);
		 	
     		// seismic fullwavefield and record
            //wavefield_output<<<grid,block>>>(u2, d_u, &d_record[k*(nx-2*pml)*(ny-2*pml)], nx, ny, nz, sz, pml);	
	 		
			 hipMemcpy(h_u2, u2, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);
			//打印 h_u2
			// for(int i=0;i<nz;i++){
			// 	for(int j=0;j<nx;j++){
			// 		for(int k=0;k<ny;k++){
			// 			if(h_u2[i*nx*ny+j*ny+k]>0.0001 || h_u2[i*nx*ny+j*ny+k]<-0.0001)
			// 				printf("h_u2[xxx]=%4.3f ",h_u2[i*nx*ny+j*ny+k]);
			// 		}
					
			// 	}
				
			// }

	 		for(i=pml;i<nx-pml;i++)
	 			for(j=pml;j<ny-pml;j++)
	 			{
	 				record[k*(nx-2*pml)*(ny-2*pml)+(i-pml)*(ny-2*pml)+j-pml] = h_u2[sz*nx*ny+i*ny+j];
	 			}
	 	
		
      		if(snapflag ==1 && k%ntsnap==0)
     		{
				sprintf(snapname,"%s%d.dat", snap_file, k);
				sprintf(snapxzname,"%s_xz%d.dat", snap_file, k);
				sprintf(snapyzname,"%s_yz%d.dat", snap_file, k);
				sprintf(snapxyname,"%s_xy%d.dat", snap_file, k);
    //  	 	hipMemcpy(snap, d_u, (nz-2*pml)*(nx-2*pml)*(ny-2*pml)*sizeof(float), hipMemcpyDeviceToHost);
    		 	for(i=pml;i<nz-pml;i++)
    		 		for(j=pml;j<nx-pml;j++)
	 			    	for(l=pml;l<ny-pml;l++)
	 			   				snap[(i-pml)*(nx-2*pml)*(ny-2*pml)+(j-pml)*(ny-2*pml)+l-pml] = h_u2[i*nx*ny+j*ny+l]; 
	 				 
    		
      	 		writefile_3d(snapname, snap, nz-2*pml, nx-2*pml, ny-2*pml);
				for(i=0;i<nz-2*pml;i++)
					for(j=0;j<nx-2*pml;j++)
						for(l=0;l<ny-2*pml;l++)
						{
									if(l==(ny-2*pml-1)/2)
									{
											snapxz[i*(nx-2*pml)+j] = snap[i*(nx-2*pml)*(ny-2*pml)+j*(ny-2*pml)+l];										
									}
									
									if(j==(nx-2*pml-1)/2)
									{
											snapyz[i*(ny-2*pml)+l] = snap[i*(nx-2*pml)*(ny-2*pml)+j*(ny-2*pml)+l];									
									}
									
									if(i==(nz-2*pml-1)/2)
									{
											snapxy[j*(ny-2*pml)+l] = snap[i*(nx-2*pml)*(ny-2*pml)+j*(ny-2*pml)+l];											
									}
							
						}		
				writefile_2d(snapxzname, snapxz, nz-2*pml, nx-2*pml);
				writefile_2d(snapyzname, snapyz, nz-2*pml, ny-2*pml);
				writefile_2d(snapxyname, snapxy, nx-2*pml, ny-2*pml);
					}

			//	printf("%f\n",absMaxval(snap, nx-2*pml, ny-2*pml, nz-2*pml));
				
		/*		sprintf(snapname_S,"%s_S%d.dat", snap_file, k);
				sprintf(snapSxzname,"%s_Sxz%d.dat", snap_file, k);
				sprintf(snapSyzname,"%s_Syz%d.dat", snap_file, k);
				sprintf(snapSxyname,"%s_Sxy%d.dat", snap_file, k);
				hipMemcpy(snapS, S, nx*ny*nz*sizeof(float), hipMemcpyDeviceToHost);
      	 		writefile_3d(snapname_S, snapS, nz, nx, ny); 
				for(i=0;i<nz;i++)
					for(j=0;j<nx;j++)
						for(l=0;l<ny;l++)
						{
									if(l==(ny-1)/2)
									{
											snapSxz[i*nx+j] = snapS[i*nx*ny+j*ny+l];										
									}
									
									if(j==(nx-1)/2)
									{
											snapSyz[i*ny+l] = snapS[i*nx*ny+j*ny+l];									
									}
									
									if(i==(nz-1)/2)
									{
											snapSxy[j*ny+l] = snapS[i*nx*ny+j*ny+l];											
									}
							
						}		
				writefile_2d(snapSxzname, snapSxz, nz, nx);
				writefile_2d(snapSyzname, snapSyz, nz, ny);
				writefile_2d(snapSxyname, snapSxy, nx, ny);*/
     	//	}	
  	}
	 
//	hipMemcpy(record, d_record, nt*(nx-2*pml)*(ny-2*pml)*sizeof(float), hipMemcpyDeviceToHost);

	//free device memory
 	hipFree(d_vp);hipFree(d_epsilon);hipFree(d_delta);hipFree(d_c);hipFree(d_c2);
 	hipFree(d_dlr);hipFree(d_ddlr);hipFree(d_dtb);hipFree(d_ddtb);hipFree(d_dfb);hipFree(d_ddfb);
 	hipFree(d_source);hipFree(S);hipFree(u1);hipFree(u2);hipFree(u3);
	hipFree(ux);hipFree(uy);hipFree(uz);hipFree(d_record);hipFree(d_u);

 	hipFree(wl11);hipFree(wl12);hipFree(wl13);hipFree(wl21);
 	hipFree(wl31);hipFree(wl32);hipFree(wl33);hipFree(pl1);
 	hipFree(pl2);hipFree(pl3);
 	hipFree(wr11);hipFree(wr12);hipFree(wr13);hipFree(wr21);
 	hipFree(wr31);hipFree(wr32);hipFree(wr33);hipFree(pr1);
 	hipFree(pr2);hipFree(pr3);
 	hipFree(wt11);hipFree(wt12);hipFree(wt13);hipFree(wt21);
 	hipFree(wt31);hipFree(wt32);hipFree(wt33);hipFree(pt1);
 	hipFree(pt2);hipFree(pt3);
 	hipFree(wb11);hipFree(wb12);hipFree(wb13);hipFree(wb21);
 	hipFree(wb31);hipFree(wb32);hipFree(wb33);hipFree(pb1);
 	hipFree(pb2);hipFree(pb3);
	hipFree(wf11);hipFree(wf12);hipFree(wf13);hipFree(wf21);
 	hipFree(wf31);hipFree(wf32);hipFree(wf33);hipFree(pf1);
 	hipFree(pf2);hipFree(pf3);
 	hipFree(wba11);hipFree(wba12);hipFree(wba13);hipFree(wba21);
 	hipFree(wba31);hipFree(wba32);hipFree(wba33);hipFree(pba1);
 	hipFree(pba2);hipFree(pba3);

	free(h_ux); free(h_uy); free(h_uz); free(snap); free(snapxz);free(snapyz);free(snapxy);
	free(snapS);free(snapSxz);free(snapSyz);free(snapSxy);
	
	endtime = clock();
	timespent=(float)(endtime-starttime)/CLOCKS_PER_SEC;
	printf("Singshot modeling  time-assuming is %f s.\n",timespent);

}
